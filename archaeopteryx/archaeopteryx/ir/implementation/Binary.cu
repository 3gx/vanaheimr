#include "hip/hip_runtime.h"
/*! \file   Binary.cpp
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date   Friday September 9, 2011
	\brief  The source file the IR Binary class.
*/

// Archaeopteryx Includes
#include <archaeopteryx/ir/interface/Binary.h>

#include <archaeopteryx/util/interface/File.h>

#include <archaeopteryx/util/interface/debug.h>
#include <archaeopteryx/util/interface/cstring.h>

// Vanaheimr Includes
#include <vanaheimr/asm/interface/Instruction.h>


#ifdef REPORT_BASE
#undef REPORT_BASE
#endif

#define REPORT_BASE 1

namespace archaeopteryx
{

namespace ir
{

__device__ Binary::Binary(const char* filename)
: _file(0), _ownedFile(0)
{
	_ownedFile = new util::File(filename, "r");
	
	_file = _ownedFile;

	_loadHeader();
}

__device__ Binary::Binary(File* file)
: _file(file), _ownedFile(0)
{
	_loadHeader();
}

__device__ Binary::~Binary()
{
	for(unsigned int c = 0; c != _header.codePages; ++c)
	{
		delete[] _codeSection[c];
	}
	
	for(unsigned int d = 0; d != _header.dataPages; ++d)
	{
		delete[] _dataSection[d];
	}
	
	for(unsigned int s = 0; s != _header.stringPages; ++s)
	{
		delete[] _stringSection[s];
	}
	
	delete[] _symbolTable;
	delete[] _codeSection;
	delete[] _dataSection;
	delete[] _stringSection;
	
	delete _ownedFile;
}

__device__ Binary::SymbolTableEntry* Binary::findSymbol(const char* name)
{
	_loadSymbolTable();
	
	for(unsigned int i = 0; i < _header.symbols; ++i)
	{
		SymbolTableEntry* symbol = _symbolTable + i;
			
		if(_strcmp(symbol->stringOffset, name) != 0)
		{
			return symbol;
		}
	}
	
	return 0;
}

__device__ void Binary::copyCode(InstructionContainer* code, PC pc,
	unsigned int instructions)
{
	const size_t instructionsPerPage = sizeof(PageDataType) /
		sizeof(InstructionContainer);
	
	size_t page       = pc / instructionsPerPage;
	size_t pageOffset = pc % instructionsPerPage;
	
	device_report("Copying %d instructions at PC %d\n", instructions, pc);

	while(instructions > 0)
	{
		size_t instructionsInThisPage =
			util::min((size_t)(instructionsPerPage - pageOffset),
				(size_t)instructions);
	
		device_report(" copying %d instructions from page %d\n", 
			(int)instructionsInThisPage, (int)page);
		PageDataType* pageData = getCodePage(code_begin() + page);
		device_assert(pageData != 0);

		InstructionContainer* container =
			reinterpret_cast<InstructionContainer*>(pageData);
	
		util::memcpy(code, container + pageOffset,
			sizeof(InstructionContainer) * instructionsInThisPage);
	
		instructions -= instructionsInThisPage;
		pageOffset    = 0;
		page         += 1;

		device_report("  %d instructions are remaining\n", instructions);
	}
}

__device__ bool Binary::containsFunction(const char* name)
{
	SymbolTableEntry* symbol = findSymbol(name);
	
	if(symbol == 0) return false;
	
	return symbol->type == SymbolTableEntry::FunctionType;
}

__device__ void Binary::findFunction(page_iterator& page, unsigned int& offset,
	const char* name)
{
	SymbolTableEntry* symbol = findSymbol(name);
	
	if(symbol == 0)
	{
		page   = 0;
		offset = 0;
		
		return;
	}
	
	device_assert(symbol->type == SymbolTableEntry::FunctionType);
	
	page   = code_begin() + _getCodePageId(symbol->offset);
	offset = _getCodePageOffset(symbol->offset);
}

__device__ void Binary::findVariable(page_iterator& page, unsigned int& offset,
	const char* name)
{
	SymbolTableEntry* symbol = findSymbol(name);
	
	if(symbol == 0)
	{
		page   = 0;
		offset = 0;
		
		return;
	}
	
	device_assert(symbol->type == SymbolTableEntry::VariableType);
	
	page   = data_begin() + _getDataPageId(symbol->offset);
	offset = _getDataPageOffset(symbol->offset);
}

__device__ util::string Binary::getSymbolDataAsString(const char* symbolName)
{
	device_report("   getting data for symbol '%s'\n", symbolName);
	
	SymbolTableEntry* symbol = findSymbol(symbolName);
	
	device_assert(symbol != 0);

	device_assert(symbol->type == SymbolTableEntry::VariableType);
	
	util::string result(symbol->size, '\0');
	
	_strcpy((char*)result.data(), symbol->offset);
	
	return result;
}

__device__ Binary::PC Binary::findFunctionsPC(const char* name)
{
	page_iterator page  = 0;
	unsigned int offset = 0;

	findFunction(page, offset, name);
	
	const size_t instructionsPerPage = sizeof(PageDataType) /
		sizeof(InstructionContainer);
	
	return instructionsPerPage * (page - code_begin()) + offset;
}

__device__ Binary::page_iterator Binary::code_begin()
{
	return _codeSection;
}

__device__ Binary::page_iterator Binary::code_end()
{
	return _codeSection + _header.codePages;
}

__device__ Binary::page_iterator Binary::data_begin()
{
	return _dataSection;
}

__device__ Binary::page_iterator Binary::data_end()
{
	return _dataSection + _header.dataPages;
}

__device__ Binary::page_iterator Binary::string_begin()
{
	return _stringSection;
}

__device__ Binary::page_iterator Binary::string_end()
{
	return _stringSection + _header.stringPages;
}

__device__ Binary::PageDataType* Binary::getCodePage(page_iterator page)
{
	if(*page == 0)
	{
		// TODO lock the page
		
		size_t offset = _getCodePageOffset(page);

		device_report("Loading code page (%p) at offset (%p) now...\n",
			page, offset);

		_file->seekg(offset);
		*page = (PageDataType*)new PageDataType;
		_file->read(*page, sizeof(PageDataType));
	}
	
	return *page;
}

__device__ Binary::PageDataType* Binary::getDataPage(page_iterator page)
{
	if(*page == 0)
	{
		// TODO lock the page
	
		size_t offset = _getDataPageOffset(page);

		device_report("Loading data page (%p) at offset (%p) now...\n",
			page, offset);

		_file->seekg(offset);
		*page = (PageDataType*)new PageDataType;
		_file->read(*page, sizeof(PageDataType));
	}
	
	return *page;
}

__device__ Binary::PageDataType* Binary::getStringPage(page_iterator page)
{
	device_assert(page < string_end());

	if(*page == 0)
	{
		// TODO lock the page
	
		size_t offset = _getStringPageOffset(page);

		device_report("Loading string page (%p) at offset (%p) now...\n",
			page, offset);

		_file->seekg(offset);
		*page = (PageDataType*)new PageDataType;
		_file->read(*page, sizeof(PageDataType));
	}
	
	return *page;
}


__device__ void Binary::_loadHeader()
{
	_file->read(&_header, sizeof(Header));
	
	device_report("Loading header (%p magic)\n", _header.magic);
	
	device_assert(_header.magic == Header::MagicNumber);
	
	_dataSection   = new PagePointer[_header.dataPages];
	_codeSection   = new PagePointer[_header.codePages];
	_stringSection = new PagePointer[_header.stringPages];

	_symbolTable = 0;

	util::memset(_dataSection,   0, _header.dataPages   * sizeof(PagePointer));
	util::memset(_codeSection,   0, _header.codePages   * sizeof(PagePointer));
	util::memset(_stringSection, 0, _header.stringPages * sizeof(PagePointer));
	
	device_report("Loaded binary (%d data pages, %d code pages, "
		"%d symbols, %d string pages)\n", _header.dataPages, _header.codePages,
		_header.symbols, _header.stringPages);
}

__device__ void Binary::_loadSymbolTable()
{
	if(_header.symbols == 0) return;
	if(_symbolTable != 0)    return;

	device_report(" Loading symbol table now.\n");

	_symbolTable = new SymbolTableEntry[_header.symbols];
	
	device_report("  symbol table offset %d.\n", (int)_header.symbolOffset);
	device_assert(_file != 0);

	_file->seekg(_header.symbolOffset);

	device_report("  loading symbol table now.\n");

	_file->read(_symbolTable, _header.symbols * sizeof(SymbolTableEntry));

	device_report("   loaded %d symbols...\n", _header.symbols);
}

__device__ size_t Binary::_getCodePageOffset(page_iterator page)
{
	return _header.codeOffset +	(page - code_begin()) * sizeof(PageDataType);
}

__device__ size_t Binary::_getDataPageOffset(page_iterator page)
{
	return _header.dataOffset + (page - data_begin()) * sizeof(PageDataType);
}

__device__ size_t Binary::_getStringPageOffset(page_iterator page)
{
	return _header.stringsOffset +
		(page - string_begin()) * sizeof(PageDataType);
}

__device__ int Binary::_strcmp(unsigned int stringTableOffset,
	const char* string)
{
	page_iterator page  = string_begin() + _getStringPageId(stringTableOffset);
	unsigned int offset = _getStringPageOffset(stringTableOffset);

	device_report("comparing string at offset %d against '%s'\n",
		stringTableOffset, string);
	
	for(; page != string_end(); ++page, offset = 0)
	{
		const char* data = (const char*)*getStringPage(page);
		
		for(; offset != sizeof(PageDataType); ++offset, ++string)
		{
			if(data[offset] != *string)
			{
				return -1;
			}
			
			if(data[offset] == '\0')
			{
				if(*string == '\0')
				{
					return 0;
				}
				
				return -1;
			}
			else if(*string == '\n')
			{
				return -1;
			}
		}
	}
	
	return 0;
}

__device__ void Binary::_strcpy(char* string, unsigned int stringTableOffset)
{
	page_iterator page  = string_begin() + _getStringPageId(stringTableOffset);
	unsigned int offset = _getStringPageOffset(stringTableOffset);

	for(; page != string_end(); ++page, offset = 0)
	{
		const char* data = (const char*)*getStringPage(page);
		
		for(; offset != sizeof(PageDataType); ++offset, ++string)
		{
			if(data[offset] == '\0')
			{
				return;
			}
			
			*string = data[offset];
		}
	}
}

__device__ unsigned int Binary::_getCodePageId(size_t offset)
{
	size_t codeOffset = offset - _header.codeOffset;
	
	return codeOffset / sizeof(PageDataType);
}

__device__ unsigned int Binary::_getCodePageOffset(size_t offset)
{
	size_t codeOffset = offset - _header.codeOffset;
	
	return codeOffset % sizeof(PageDataType);
}

__device__ unsigned int Binary::_getDataPageId(size_t offset)
{
	size_t dataOffset = offset - _header.dataOffset;
	
	return dataOffset / sizeof(PageDataType);
}

__device__ unsigned int Binary::_getDataPageOffset(size_t offset)
{
	size_t dataOffset = offset - _header.dataOffset;
	
	return dataOffset % sizeof(PageDataType);
}

__device__ unsigned int Binary::_getStringPageId(size_t offset)
{
	device_assert(offset >= _header.stringsOffset);

	size_t stringsOffset = offset - _header.stringsOffset;
	
	return stringsOffset / sizeof(PageDataType);
}

__device__ unsigned int Binary::_getStringPageOffset(size_t offset)
{
	size_t stringsOffset = offset - _header.stringsOffset;
	
	return stringsOffset % sizeof(PageDataType);
}

}

}

