#include "hip/hip_runtime.h"
/*	\file   HostReflection.cpp
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date   Saturday July 16, 2011
	\brief  The source file for the HostReflection set of functions.
*/

// Archaeopteryx Includes
#include <archaeopteryx/util/interface/HostReflection.h>
#include <archaeopteryx/util/interface/ThreadId.h>
#include <archaeopteryx/util/interface/cstring.h>
#include <archaeopteryx/util/interface/StlFunctions.h>
#include <archaeopteryx/util/interface/debug.h>

// Standard Library Includes
#include <cstring>
#include <cassert>
#include <fstream>



// Forward Declarations

namespace ocelot
{
	void launch(const std::string& moduleName, const std::string& kernelName);
}

// Preprocessor Macros
#ifdef REPORT_BASE
#undef REPORT_BASE
#endif

#define REPORT_BASE 1

namespace archaeopteryx
{

namespace util
{

// TODO Remove these when __device__ can be embedded in a clas
__device__ HostReflection::DeviceQueue* _hostToDevice;
__device__ HostReflection::DeviceQueue* _deviceToHost;

template <typename T>
__device__ T HostReflection::Payload::get(unsigned int i)
{
	T temp = 0;
	
	std::memcpy(&temp, data.data + data.indexes[i], sizeof(T));

	return temp;
}

__device__ HostReflection::KernelLaunchMessage::KernelLaunchMessage(
	unsigned int ctas, unsigned int threads,
	const char* name, const Payload& payload)
: _stringLength(util::strlen(name) + 1), _data(new char[payloadSize()])
{
	char* data = _data;
	
	std::memcpy(data, &payload.data, sizeof(PayloadData));
	data += sizeof(PayloadData);

	std::memcpy(data, &ctas, sizeof(unsigned int));
	data += sizeof(unsigned int);

	std::memcpy(data, &threads, sizeof(unsigned int));
	data += sizeof(unsigned int);
	
	std::memcpy(data, &_stringLength, sizeof(unsigned int));
	data += sizeof(unsigned int);
	
	std::memcpy(data, name, _stringLength);
	data += _stringLength;
}

__device__ HostReflection::KernelLaunchMessage::~KernelLaunchMessage()
{
	delete[] _data;
}

__device__ void* HostReflection::KernelLaunchMessage::payload() const
{
	return _data;
}

__device__ size_t HostReflection::KernelLaunchMessage::payloadSize() const
{
	return sizeof(unsigned int) * 3 + sizeof(Payload) + _stringLength;
}

__device__ HostReflection::HandlerId
	HostReflection::KernelLaunchMessage::handler() const
{
	return KernelLaunchMessageHandler;
}

__device__ void HostReflection::sendAsynchronous(const Message& m)
{
	unsigned int bytes = m.payloadSize() + sizeof(Header);

	char* buffer = new char[bytes];
	
	Header* header = reinterpret_cast<Header*>(buffer);
	
	header->type     = Asynchronous;
	header->threadId = threadIdx.x;
	header->size     = bytes;
	header->handler  = m.handler();
	
	std::memcpy(buffer + sizeof(Header), m.payload(), m.payloadSize());
	 
	device_report(" sending asynchronous gpu->host message "
		"(%d type, %d id, %d size, %d handler)\n", Asynchronous,	
		header->threadId, bytes, m.handler());
	
	while(!_deviceToHost->push(buffer, bytes));

	delete[] buffer;
}

__device__ void HostReflection::sendSynchronous(const Message& m)
{
	unsigned int bytes = m.payloadSize() + sizeof(SynchronousHeader);

	char* buffer = new char[bytes];
	
	SynchronousHeader* header = reinterpret_cast<SynchronousHeader*>(buffer);
	
	header->type     = Synchronous;
	header->threadId = threadIdx.x;
	header->size     = bytes;
	header->handler  = m.handler();
	
	volatile bool* flag = new bool;
	*flag = false;

	header->address = (void*)flag;	

	std::memcpy(buffer + sizeof(SynchronousHeader), m.payload(),
		m.payloadSize());
	 
	device_report(" sending synchronous gpu->host message "
		"(%d type, %d id, %d size, %d handler, %x flag)\n", Synchronous,	
		header->threadId, bytes, m.handler(), header->address);
	
	while(!_deviceToHost->push(buffer, bytes));

	device_report("  waiting for ack...\n");
	
	while(*flag == false);

	device_report("   ...received ack\n");
	
	delete flag;
	delete[] buffer;
}

__device__ void HostReflection::receive(Message& m)
{
	while(!_hostToDevice->peek());

	device_report(" receiving cpu->gpu message.");

	size_t bytes = m.payloadSize() + sizeof(Header);

	char* buffer = new char[bytes];
	
	_hostToDevice->pull(buffer, bytes);

	device_report("  bytes: %d\n", (int)(bytes - sizeof(Header)));

	std::memcpy(m.payload(), (buffer + sizeof(Header)), m.payloadSize());

	delete[] buffer;
}

__device__ void HostReflection::launch(unsigned int ctas, unsigned int threads,
	const char* functionName, const Payload& payload)
{
	KernelLaunchMessage message(ctas, threads, functionName, payload);

	sendAsynchronous(message);
}

__device__ unsigned int align(unsigned int address, unsigned int alignment)
{
	unsigned int remainder = address % alignment;
	return remainder == 0 ? address : address + (alignment - remainder);
}

template<typename T0, typename T1, typename T2, typename T3, typename T4>
__device__ HostReflection::Payload HostReflection::createPayload(const T0& t0,
	const T1& t1, const T2& t2, const T3& t3, const T4& t4)
{
	Payload result;

	PayloadData& payload = result.data;

	unsigned int index = 0;
	
	payload.indexes[0] = index;
	std::memcpy(payload.data + index, &t0, sizeof(T0));
	index += sizeof(T0);
	index =  align(index, sizeof(T1));
	
	payload.indexes[1] = index;
	std::memcpy(payload.data + index, &t1, sizeof(T1));
	index += sizeof(T1);
	index =  align(index, sizeof(T2));
	
	payload.indexes[2] = index;
	std::memcpy(payload.data + index, &t2, sizeof(T2));
	index += sizeof(T2);
	index =  align(index, sizeof(T3));
	
	payload.indexes[3] = index;
	std::memcpy(payload.data + index, &t3, sizeof(T3));
	index += sizeof(T3);
	index =  align(index, sizeof(T4));
	
	payload.indexes[4] = index;
	std::memcpy(payload.data + index, &t4, sizeof(T4));

	return result;
}

template<typename T0, typename T1, typename T2, typename T3>
__device__ HostReflection::Payload HostReflection::createPayload(const T0& t0,
	const T1& t1, const T2& t2, const T3& t3)
{
	return createPayload(t0, t1, t2, t3, (int)0);
}

template<typename T0, typename T1, typename T2>
__device__ HostReflection::Payload HostReflection::createPayload(const T0& t0,
	const T1& t1, const T2& t2)
{
	return createPayload(t0, t1, t2, (int)0);
}

template<typename T0, typename T1>
__device__ HostReflection::Payload HostReflection::createPayload(const T0& t0,
	const T1& t1)
{
	return createPayload(t0, t1, (int)0);
}

template<typename T0>
__device__ HostReflection::Payload HostReflection::createPayload(const T0& t0)
{
	return createPayload(t0, (int)0);
}

__device__ HostReflection::Payload HostReflection::createPayload()
{
	return createPayload((int)0);
}

__device__ size_t HostReflection::maxMessageSize()
{
	return 512;
}

__device__ HostReflection::DeviceQueue::DeviceQueue(QueueMetaData* m)
: _metadata(m)
{
	device_report("binding device queue to metadata (%d size, "
		"%d head, %d tail, %d mutex)\n", (int)m->size, (int)m->head,
		(int)m->tail, m->mutex);
}

__device__ HostReflection::DeviceQueue::~DeviceQueue()
{

}

__device__ bool HostReflection::DeviceQueue::push(const void* data, size_t size)
{
	device_assert(size <= this->size());

	if(size > _capacity()) return false;
	
	if(!_lock()) return false;	

	device_report("pushing %d bytes into gpu->cpu queue.\n", (int)size);

	size_t end  = _metadata->size;
	size_t head = _metadata->head;

	size_t remainder = end - head;
	size_t firstCopy = min(remainder, size);

	std::memcpy(_metadata->deviceBegin + head, data, firstCopy);

	bool secondCopyNecessary = firstCopy != size;

	size_t secondCopy = secondCopyNecessary ? size - firstCopy : 0;
	
	std::memcpy(_metadata->deviceBegin, (char*)data + firstCopy, secondCopy);
	_metadata->head = secondCopyNecessary ? secondCopy : head + firstCopy;
	
	device_report(" after push (%d used, %d remaining, %d size)\n",
		(int)_used(), (int)_capacity(), (int)this->size());
	
	_unlock();
	
	return true;
}

__device__ bool HostReflection::DeviceQueue::pull(void* data, size_t size)
{
	device_assert(size <= _used());

	if(!_lock()) return false;
	
	_metadata->tail = _read(data, size);

	_unlock();
	
	return true;
}

__device__ bool HostReflection::DeviceQueue::peek()
{
	if(_used() < sizeof(Header)) return false;

	if(!_lock()) return false;
	
	Header header;
	
	_read(&header, sizeof(Header));
	
	_unlock();
	
	return header.threadId == threadId();
}

__device__ size_t HostReflection::DeviceQueue::size() const
{
	return _metadata->size;
}

__device__  size_t HostReflection::DeviceQueue::_used() const
{
	size_t end  = _metadata->size;
	size_t head = _metadata->head;
	size_t tail = _metadata->tail;
	
	size_t greaterOrEqual = head - tail;
	size_t less           = (head) + (end - tail);
	
	bool isGreaterOrEqual = head >= tail;
	
	return (isGreaterOrEqual) ? greaterOrEqual : less;
}

__device__  size_t HostReflection::DeviceQueue::_capacity() const
{
	return size() - _used();
}

__device__ bool HostReflection::DeviceQueue::_lock()
{
	device_assert(_metadata->mutex != threadId());
	
	size_t result = atomicCAS((long long unsigned int*)&_metadata->mutex,
		(long long unsigned int)-1, (long long unsigned int)threadId());
	
	return result == (size_t)-1;
}

__device__ void HostReflection::DeviceQueue::_unlock()
{
	device_assert(_metadata->mutex == threadId());
	
	_metadata->mutex = (size_t)-1;
}

__device__ size_t HostReflection::DeviceQueue::_read(
	void* data, size_t size)
{
	size_t end  = _metadata->size;
	size_t tail = _metadata->tail;

	size_t remainder = end - tail;
	size_t firstCopy = min(remainder, size);

	std::memcpy(data, _metadata->deviceBegin + tail, firstCopy);

	bool secondCopyNecessary = firstCopy != size;

	size_t secondCopy = secondCopyNecessary ? size - firstCopy : 0;
	
	std::memcpy((char*)data + firstCopy, _metadata->deviceBegin, secondCopy);
	
	return secondCopyNecessary ? secondCopy : tail + firstCopy;
}

__global__ void _bootupHostReflection(
	HostReflection::QueueMetaData* hostToDeviceMetadata,
	HostReflection::QueueMetaData* deviceToHostMetadata)
{
	_hostToDevice = new HostReflection::DeviceQueue(hostToDeviceMetadata);
	_deviceToHost = new HostReflection::DeviceQueue(deviceToHostMetadata);
}

__global__ void _teardownHostReflection()
{
	delete _hostToDevice;
	delete _deviceToHost;
}

}

}

